#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <stdio.h>

using namespace std;


__global__ void DijkstraAlgo(int* graph, int src, int* res, bool* Tset, int size)
{

	for (int k = 0; k < size; k++)
	{
		
		int minimum = INT_MAX, ind;

		for (int m = 0; m < size; m++)
		{
			if (Tset[m] == false && res[m] <= minimum)
			{
				minimum = res[m];
				ind = m;
			}
		}
		int m = ind;
		Tset[m] = true;
		for (int j = 0; j < size; j++)
		{

			if (!Tset[j] && graph[m* size +j] && res[m] != INT_MAX && res[m] + graph[m* size +j] < res[j])
				res[j] = res[m] + graph[m* size +j];
		}
	}

}

int main()
{
	const int src = 0;
	const int size = 6;
	const int memSize = sizeof(int) * size * size;
	int graph[size][size] = {
		{0, 1, 2, 0, 0, 0},
		{1, 0, 0, 5, 1, 0},
		{2, 0, 0, 2, 3, 0},
		{0, 5, 2, 0, 2, 2},
		{0, 1, 3, 2, 0, 1},
		{0, 0, 0, 2, 1, 0} };

	int distance[size];
	bool Tset[size];
	for (int i= 0; i < size; i++) {
		distance[i] = INT_MAX;
		Tset[i] = false;
	}
	distance[src] = 0;

	int* gpu_graph, * gpu_distance;
	bool *gpuTset;
	hipMalloc((void**)&gpu_graph, memSize);
	hipMalloc((void**)&gpu_distance, sizeof(int)* size);
	hipMalloc((void**)&gpuTset, sizeof(int) * size);

	hipMemcpy(gpu_graph, graph, memSize, hipMemcpyKind::hipMemcpyHostToDevice);
	hipMemcpy(gpu_distance, distance, sizeof(int)*size, hipMemcpyKind::hipMemcpyHostToDevice);
	hipMemcpy(gpuTset, Tset, sizeof(int) * size, hipMemcpyKind::hipMemcpyHostToDevice);

	DijkstraAlgo << <1, 1 >> > (gpu_graph, 0, gpu_distance, gpuTset, size);
	cout << "Vertex\t\tDistance from source vertex" << endl;

	hipMemcpy(distance, gpu_distance, sizeof(int)*size, hipMemcpyKind::hipMemcpyDeviceToHost);
	for (int k = 0; k < size; k++)
	{
		char str = 65 + k;
		cout << str << "\t\t\t" << distance[k] << endl;
	}
	return 0;
}
